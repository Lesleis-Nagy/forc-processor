#include "hip/hip_runtime.h"
//
// Created by L. Nagy on 21/07/2023.
//

#include <chrono>
#include <iostream>


#include "rbf_cuda_interp.hpp"

__global__
void saxpy(Real a, const Real *x, Real *y) {

    size_t i = blockIdx.x*blockDim.x + threadIdx.x;
    y[i] = a*x[i] + y[i];

}

void cuda_saxpy(Real a,
                const std::vector<Real> &in_x,
                const std::vector<Real> &in_y,
                std::vector<Real> &out) {

    Real *d_x, *d_y;

    size_t threadsPerBlock = 256;
    size_t blocksPerGrid = (in_x.size() + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory on CUDA device.
    std::cout << "Allocating memory on device ...\n";
    auto start_cudaMalloc = std::chrono::high_resolution_clock::now();
    hipMalloc(&d_x, in_x.size()*sizeof(Real));
    hipMalloc(&d_y, in_y.size()*sizeof(Real));
    auto stop_cudaMalloc = std::chrono::high_resolution_clock::now();
    auto duration_cudaMalloc = std::chrono::duration_cast<std::chrono::microseconds>(stop_cudaMalloc - start_cudaMalloc);
    std::cout << "OK! (" << duration_cudaMalloc.count() << " us)\n";

    // Copy x and y arrays to device.
    std::cout << "Copying memory to device ...\n";
    auto start_cudaMemcpy1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_x, in_x.data(), in_x.size()*sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_y, in_y.data(), in_y.size()*sizeof(Real), hipMemcpyHostToDevice);
    auto stop_cudaMemcpy1 = std::chrono::high_resolution_clock::now();
    auto duration_cudaMemcpy1 = std::chrono::duration_cast<std::chrono::microseconds>(stop_cudaMemcpy1 - start_cudaMemcpy1);
    std::cout << "OK! (" << duration_cudaMemcpy1.count() << " us)\n";

    out.resize(in_x.size());

    // Perform SAXPY.
    std::cout << "Running saxpy kernel ...\n";
    auto start_saxpy = std::chrono::high_resolution_clock::now();
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(a, d_x, d_y);
    auto stop_saxpy = std::chrono::high_resolution_clock::now();
    auto duration_saxpy = std::chrono::duration_cast<std::chrono::microseconds>(stop_saxpy - start_saxpy);
    std::cout << "OK! (" << duration_saxpy.count() << " us)\n";

    std::cout << "Copying memory to CPU ...\n";
    auto start_cudaMemcpy2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(out.data(), d_y, out.size() * sizeof(Real), hipMemcpyDeviceToHost);
    auto stop_cudaMemcpy2 = std::chrono::high_resolution_clock::now();
    auto duration_cudaMemcpy2 = std::chrono::duration_cast<std::chrono::microseconds>(stop_cudaMemcpy2 - start_cudaMemcpy2);
    std::cout << "OK! (" << duration_cudaMemcpy2.count() << " us)\n";

    hipFree(d_x);
    hipFree(d_y);

}
