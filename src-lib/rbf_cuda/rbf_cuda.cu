#include "hip/hip_runtime.h"
//
// Created by L. Nagy on 21/07/2023.
//

#include "rbf_cuda.h"

__global__
void saxpy(float a, float *x, float *y) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    y[i] = a*x[i] + y[i];

}

void cuda_saxpy(float a, float *in_x, float *in_y, float *out_x, int n) {

    float *d_x, *d_y;

    // Allocate memory on CUDA device.
    hipMalloc(&d_x, n*sizeof(float));
    hipMalloc(&d_y, n*sizeof(float));

    // Copy x and y arrays to device.
    hipMemcpy(d_x, in_x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, in_y, n*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<1, n>>>(a, d_x, d_y);

    hipMemcpy(out_x, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

}
