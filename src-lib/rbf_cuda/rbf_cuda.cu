#include "hip/hip_runtime.h"
//
// Created by L. Nagy on 21/07/2023.
//

#include "rbf_cuda.hpp"

__global__
void saxpy(float a, float *x, float *y) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    y[i] = a*x[i] + y[i];

}

void cuda_saxpy(float a,
                const std::vector<float> &in_x,
                const std::vector<float> &in_y,
                std::vector<float> &out) {

    float *d_x, *d_y;

    // Allocate memory on CUDA device.
    hipMalloc(&d_x, in_x.size()*sizeof(float));
    hipMalloc(&d_y, in_y.size()*sizeof(float));

    // Copy x and y arrays to device.
    hipMemcpy(d_x, in_x.data(), in_x.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, in_y.data(), in_y.size()*sizeof(float), hipMemcpyHostToDevice);

    out.resize(in_x.size());

    // Perform SAXPY on 1M elements
    saxpy<<<1, out.size()>>>(a, d_x, d_y);

    hipMemcpy(out.data(), d_y, out.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

}
